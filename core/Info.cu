#include "hip/hip_runtime.h"
// Dear emacs, this is -*- c++ -*-

// Local include(s).
#include "Info.h"
#include "Macros.cuh"

// System include(s).
#include <cmath>

Info& Info::instance() {

   static Info info;
   return info;
}

int Info::nDevices() const {

   return m_nDevices;
}

const std::vector< std::string >& Info::names() const {

   return m_names;
}

const std::vector< int >& Info::maxThreadsPerBlock() const {

   return m_maxThreadsPerBlock;
}

const std::vector< bool >& Info::concurrentKernels() const {

   return m_concurrentKernels;
}

const std::vector< std::size_t >& Info::totalMemory() const {

   return m_totalMemory;
}

void Info::print() const {

   std::cout << *this << std::endl;
   return;
}

Info::Info() {

   // Collect all information about all the available devices on
   // construction.
   CUDA_IGNORE( hipGetDeviceCount( &m_nDevices ) );
   for( int i = 0; i < m_nDevices; ++i ) {

      // Retrieve all properties of this device.
      hipDeviceProp_t properties;
      CUDA_CHECK( hipGetDeviceProperties( &properties, i ) );

      // Set the values in our variables.
      m_names.push_back( properties.name );
      m_maxThreadsPerBlock.push_back( properties.maxThreadsPerBlock );
      m_concurrentKernels.push_back( properties.concurrentKernels );
      m_totalMemory.push_back( properties.totalGlobalMem );
   }
}

namespace std {

   std::ostream& operator<< ( std::ostream& out, const Info& info ) {

      if( info.nDevices() > 0 ) {
         for( int i = 0; i < info.nDevices(); ++i ) {
            out << " /-- Device ID " << i << " " << std::string( 31, '-' )
                << "\\" << std::endl;
            out << " | Name: " << info.names()[ i ]
                << std::string( ( 39 > info.names()[ i ].length() ?
                                  39 - info.names()[ i ].length() : 0 ), ' ' )
                << "|" << std::endl;
            const std::size_t threadDigits =
               ( static_cast< std::size_t >( std::log10( info.maxThreadsPerBlock()[ i ] ) )
                 + 1 );
            out << " | Max. threads per block: "
                << info.maxThreadsPerBlock()[ i ]
                << std::string( ( 21 > threadDigits ?
                                  21 - threadDigits : 0 ), ' ' )
                << "|" << std::endl;
            out << " | Concurrent kernels: "
                << ( info.concurrentKernels()[ i ] ? "true " : "false" )
                << std::string( 20, ' ' ) << "|" << std::endl;
            static const double MegaBytes = 1.0 / ( 1024 * 1024 );
            const double totalMem = info.totalMemory()[ i ] * MegaBytes;
            const std::size_t memDigits =
               ( static_cast< std::size_t >( std::log10( totalMem ) ) + 1 );
            out << " | Total memory: " << totalMem << " MB"
                << std::string( ( 25 > memDigits ?
                                  25 - memDigits : 0 ), ' ' ) << "|"
                << std::endl;
            out << " \\" << std::string( 46, '-' ) << "/";
         }
      } else {
         out << "<No CUDA device(s)>";
      }
      return out;
   }

} // namespace std
